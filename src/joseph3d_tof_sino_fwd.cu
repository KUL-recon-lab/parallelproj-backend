#include "hip/hip_runtime.h"
#include "parallelproj.h"
#include "joseph3d_tof_sino_fwd_worker.h"
#include "debug.h"
#include "cuda_utils.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

__global__ void joseph3d_tof_sino_fwd_kernel(const float *xstart,
                                             const float *xend,
                                             const float *img,
                                             const float *img_origin,
                                             const float *voxsize,
                                             float *p,
                                             size_t nlors,
                                             const int *img_dim,
                                             float tofbin_width,
                                             const float *sigma_tof,
                                             const float *tofcenter_offset,
                                             float n_sigmas,
                                             short n_tofbins,
                                             unsigned char lor_dependent_sigma_tof,
                                             unsigned char lor_dependent_tofcenter_offset)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nlors)
    {
        joseph3d_tof_sino_fwd_worker(i, xstart, xend, img, img_origin, voxsize, p, img_dim, tofbin_width,
                                     sigma_tof, tofcenter_offset, n_sigmas, n_tofbins,
                                     lor_dependent_sigma_tof, lor_dependent_tofcenter_offset);
    }
}

//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////

void joseph3d_tof_sino_fwd(const float *xstart,
                           const float *xend,
                           const float *img,
                           const float *img_origin,
                           const float *voxsize,
                           float *p,
                           size_t nlors,
                           const int *img_dim,
                           float tofbin_width,
                           const float *sigma_tof,
                           const float *tofcenter_offset,
                           float n_sigmas,
                           short n_tofbins,
                           unsigned char lor_dependent_sigma_tof,
                           unsigned char lor_dependent_tofcenter_offset,
                           int device_id,
                           int threadsperblock)
{
    // Calculate nvoxels from img_dim - img_dim can be device pointer!
    size_t nvoxels = cuda_nvoxels_from_img_dim(img_dim);

    // Set the CUDA device
    if (device_id >= 0)
    {
        hipSetDevice(device_id);
    }

    /////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////
    // copy arrays to device if needed
    /////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////

    // Handle xstart (read mostly)
    float *d_xstart = nullptr;
    bool free_xstart = false;
    handle_cuda_input_array(xstart, &d_xstart, sizeof(float) * nlors * 3, free_xstart, device_id, hipMemAdviseSetReadMostly);

    // Handle xend (read mostly)
    float *d_xend = nullptr;
    bool free_xend = false;
    handle_cuda_input_array(xend, &d_xend, sizeof(float) * nlors * 3, free_xend, device_id, hipMemAdviseSetReadMostly);

    // Handle img (read mostly)
    float *d_img = nullptr;
    bool free_img = false;
    handle_cuda_input_array(img, &d_img, sizeof(float) * nvoxels, free_img, device_id, hipMemAdviseSetReadMostly);

    // Handle img_origin (read mostly)
    float *d_img_origin = nullptr;
    bool free_img_origin = false;
    handle_cuda_input_array(img_origin, &d_img_origin, sizeof(float) * 3, free_img_origin, device_id, hipMemAdviseSetReadMostly);

    // Handle voxsize (read mostly)
    float *d_voxsize = nullptr;
    bool free_voxsize = false;
    handle_cuda_input_array(voxsize, &d_voxsize, sizeof(float) * 3, free_voxsize, device_id, hipMemAdviseSetReadMostly);

    // Handle p (write access) - size depends on tofbins
    size_t p_size = sizeof(float) * nlors * n_tofbins;
    float *d_p = nullptr;
    bool free_p = false;
    handle_cuda_input_array(p, &d_p, p_size, free_p, device_id, hipMemAdviseSetAccessedBy);

    // Handle img_dim (read mostly)
    int *d_img_dim = nullptr;
    bool free_img_dim = false;
    handle_cuda_input_array(img_dim, &d_img_dim, sizeof(int) * 3, free_img_dim, device_id, hipMemAdviseSetReadMostly);

    // Handle sigma_tof (read mostly)
    float *d_sigma_tof = nullptr;
    bool free_sigma_tof = false;
    size_t sigma_tof_size = lor_dependent_sigma_tof ? sizeof(float) * nlors : sizeof(float);
    handle_cuda_input_array(sigma_tof, &d_sigma_tof, sigma_tof_size, free_sigma_tof, device_id, hipMemAdviseSetReadMostly);

    // Handle tofcenter_offset (read mostly)
    float *d_tofcenter_offset = nullptr;
    bool free_tofcenter_offset = false;
    size_t tofcenter_offset_size = lor_dependent_tofcenter_offset ? sizeof(float) * nlors : sizeof(float);
    handle_cuda_input_array(tofcenter_offset, &d_tofcenter_offset, tofcenter_offset_size, free_tofcenter_offset, device_id, hipMemAdviseSetReadMostly);

    ////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////
    // launch the kernel
    ////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////

#ifdef DEBUG
    // get and print the current cuda device ID
    int current_device_id;
    hipGetDevice(&current_device_id);
    DEBUG_PRINT("Using CUDA device: %d\n", current_device_id);
#endif

    int num_blocks = (int)((nlors + threadsperblock - 1) / threadsperblock);
    joseph3d_tof_sino_fwd_kernel<<<num_blocks, threadsperblock>>>(
        d_xstart, d_xend, d_img, d_img_origin, d_voxsize, d_p, nlors, d_img_dim,
        tofbin_width, d_sigma_tof, d_tofcenter_offset, n_sigmas, n_tofbins,
        lor_dependent_sigma_tof, lor_dependent_tofcenter_offset);
    hipDeviceSynchronize();

    ////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////
    // free device memory if needed
    ////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////

    // Free device memory if it was allocated
    if (free_xstart)
        hipFree(d_xstart);
    if (free_xend)
        hipFree(d_xend);
    if (free_img)
        hipFree(d_img);
    if (free_img_origin)
        hipFree(d_img_origin);
    if (free_voxsize)
        hipFree(d_voxsize);
    if (free_p)
    {
        hipMemcpy(p, d_p, p_size, hipMemcpyDeviceToHost);
        hipFree(d_p);
    }
    if (free_img_dim)
        hipFree(d_img_dim);
    if (free_sigma_tof)
        hipFree(d_sigma_tof);
    if (free_tofcenter_offset)
        hipFree(d_tofcenter_offset);
}