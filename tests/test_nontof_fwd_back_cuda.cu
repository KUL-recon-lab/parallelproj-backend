#include "hip/hip_runtime.h"
#include "parallelproj.h"
#include "file_utils.h"
#include <iostream>
#include <chrono>
#include <cmath>
#include <vector>
#include <numeric>
#include <hip/hip_runtime.h>

void test_cuda_managed_arrays(int device_id, int threadsperblock);
void test_cuda_device_arrays(int device_id, int threadsperblock);
void test_box_projection_cuda_managed_arrays(int device_id, int threadsperblock);
void test_box_projection_cuda_device_arrays(int device_id, int threadsperblock);

int main()
{
  bool all_tests_passed = true;

  int device_count;
  hipGetDeviceCount(&device_count);

  for (int i = 0; i < device_count; i++)
  {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    std::cout << "Device " << i << ": " << prop.name << "\n";

    try
    {
      std::cout << "\n--- Testing with CUDA Managed Arrays ---\n";
      test_cuda_managed_arrays(i, 64);
    }
    catch (const std::exception &e)
    {
      std::cerr << "CUDA-managed array test failed on device " << i << ": " << e.what() << "\n";
      all_tests_passed = false;
    }

    std::cout << "\n--- Testing with CUDA Device Arrays ---\n";
    try
    {
      std::cout << "\n--- Testing with CUDA Device Arrays ---\n";
      test_cuda_device_arrays(i, 64);
    }
    catch (const std::exception &e)
    {
      std::cerr << "CUDA device array test failed on device " << i << ": " << e.what() << "\n";
      all_tests_passed = false;
    }

    try
    {
      std::cout << "\n--- Testing Box Projection with CUDA Managed Arrays ---\n";
      test_box_projection_cuda_managed_arrays(i, 64);
    }
    catch (const std::exception &e)
    {
      std::cerr << "CUDA-managed box projection test failed on device " << i << ": " << e.what() << "\n";
      all_tests_passed = false;
    }

    try
    {
      std::cout << "\n--- Testing Box Projection with CUDA Device Arrays ---\n";
      test_box_projection_cuda_device_arrays(i, 64);
    }
    catch (const std::exception &e)
    {
      std::cerr << "CUDA device array box projection test failed on device " << i << ": " << e.what() << "\n";
      all_tests_passed = false;
    }
  }

  return all_tests_passed ? 0 : 1;
}

void test_cuda_managed_arrays(int device_id, int threadsperblock)
{
  hipSetDevice(device_id);

  // CUDA-managed array test
  int *cm_img_dim;
  hipMallocManaged(&cm_img_dim, 3 * sizeof(int));
  cm_img_dim[0] = 2;
  cm_img_dim[1] = 3;
  cm_img_dim[2] = 4;

  float *cm_voxsize;
  hipMallocManaged(&cm_voxsize, 3 * sizeof(float));
  cm_voxsize[0] = 4.0f;
  cm_voxsize[1] = 3.0f;
  cm_voxsize[2] = 2.0f;

  float *cm_img_origin;
  hipMallocManaged(&cm_img_origin, 3 * sizeof(float));
  for (int i = 0; i < 3; ++i)
  {
    cm_img_origin[i] = (-(float)cm_img_dim[i] / 2 + 0.5f) * cm_voxsize[i];
  }

  size_t nvoxels = cm_img_dim[0] * cm_img_dim[1] * cm_img_dim[2];

  std::vector<float> h_img = readArrayFromFile<float>("img.txt");
  float *cm_img;
  hipMallocManaged(&cm_img, h_img.size() * sizeof(float));
  std::copy(h_img.begin(), h_img.end(), cm_img);

  std::vector<float> h_vstart = readArrayFromFile<float>("vstart.txt");
  float *cm_vstart;
  hipMallocManaged(&cm_vstart, h_vstart.size() * sizeof(float));
  std::copy(h_vstart.begin(), h_vstart.end(), cm_vstart);

  std::vector<float> h_vend = readArrayFromFile<float>("vend.txt");
  float *cm_vend;
  hipMallocManaged(&cm_vend, h_vend.size() * sizeof(float));
  std::copy(h_vend.begin(), h_vend.end(), cm_vend);

  size_t nlors = h_vstart.size() / 3;

  float *cm_xstart, *cm_xend;
  hipMallocManaged(&cm_xstart, 3 * nlors * sizeof(float));
  hipMallocManaged(&cm_xend, 3 * nlors * sizeof(float));

  for (int ir = 0; ir < nlors; ir++)
  {
    for (int j = 0; j < 3; j++)
    {
      cm_xstart[ir * 3 + j] = cm_img_origin[j] + cm_vstart[ir * 3 + j] * cm_voxsize[j];
      cm_xend[ir * 3 + j] = cm_img_origin[j] + cm_vend[ir * 3 + j] * cm_voxsize[j];
    }
  }

  float *cm_img_fwd;
  hipMallocManaged(&cm_img_fwd, nlors * sizeof(float));
  joseph3d_fwd(cm_xstart, cm_xend, cm_img, cm_img_origin, cm_voxsize, cm_img_fwd, nvoxels, nlors, cm_img_dim, device_id, threadsperblock);

  std::vector<float> h_expected_fwd_vals = readArrayFromFile<float>("expected_fwd_vals.txt");
  float fwd_diff = 0;
  float eps = 1e-7;

  for (int ir = 0; ir < nlors; ir++)
  {
    fwd_diff = std::abs(cm_img_fwd[ir] - h_expected_fwd_vals[ir]);
    if (fwd_diff > eps)
    {
      std::cerr << "CUDA-managed array test failed for ray " << ir << "\n";
      return;
    }
  }

  // Test the back projection
  float *cm_bimg;
  hipMallocManaged(&cm_bimg, cm_img_dim[0] * cm_img_dim[1] * cm_img_dim[2] * sizeof(float));
  std::fill(cm_bimg, cm_bimg + (cm_img_dim[0] * cm_img_dim[1] * cm_img_dim[2]), 0.0f);

  float *cm_ones;
  hipMallocManaged(&cm_ones, nlors * sizeof(float));
  std::fill(cm_ones, cm_ones + nlors, 1.0f);

  joseph3d_back(cm_xstart, cm_xend, cm_bimg, cm_img_origin, cm_voxsize, cm_ones, nvoxels, nlors, cm_img_dim, device_id, threadsperblock);

  printf("\nCUDA-managed back projection of ones along all rays:\n");
  for (size_t i0 = 0; i0 < cm_img_dim[0]; i0++)
  {
    for (size_t i1 = 0; i1 < cm_img_dim[1]; i1++)
    {
      for (size_t i2 = 0; i2 < cm_img_dim[2]; i2++)
      {
        printf("%.1f ", cm_bimg[cm_img_dim[1] * cm_img_dim[2] * i0 + cm_img_dim[2] * i1 + i2]);
      }
      printf("\n");
    }
    printf("\n");
  }

  // Validate the back projection using adjointness
  float inner_product1 = 0.0f;
  float inner_product2 = 0.0f;

  for (size_t i = 0; i < h_img.size(); i++)
  {
    inner_product1 += cm_img[i] * cm_bimg[i];
  }

  for (size_t ir = 0; ir < nlors; ir++)
  {
    inner_product2 += cm_img_fwd[ir] * cm_ones[ir];
  }

  float ip_diff = fabs(inner_product1 - inner_product2);

  if (ip_diff > eps)
  {
    std::cerr << "CUDA-managed array back projection test failed: adjointness property violated.\n";
  }

  hipFree(cm_img_dim);
  hipFree(cm_voxsize);
  hipFree(cm_img_origin);
  hipFree(cm_img);
  hipFree(cm_vstart);
  hipFree(cm_vend);
  hipFree(cm_xstart);
  hipFree(cm_xend);
  hipFree(cm_img_fwd);
  hipFree(cm_bimg);
  hipFree(cm_ones);
}

void test_cuda_device_arrays(int device_id, int threadsperblock)
{
  hipSetDevice(device_id);

  // CUDA device array test
  int *d_img_dim;
  hipMalloc(&d_img_dim, 3 * sizeof(int));
  int h_img_dim[3] = {2, 3, 4};
  size_t nvoxels = h_img_dim[0] * h_img_dim[1] * h_img_dim[2];
  hipMemcpy(d_img_dim, h_img_dim, 3 * sizeof(int), hipMemcpyHostToDevice);

  float *d_voxsize;
  hipMalloc(&d_voxsize, 3 * sizeof(float));
  float h_voxsize[3] = {4.0f, 3.0f, 2.0f};
  hipMemcpy(d_voxsize, h_voxsize, 3 * sizeof(float), hipMemcpyHostToDevice);

  float h_img_origin[3];
  for (int i = 0; i < 3; ++i)
  {
    h_img_origin[i] = (-(float)h_img_dim[i] / 2 + 0.5f) * h_voxsize[i];
  }
  float *d_img_origin;
  hipMalloc(&d_img_origin, 3 * sizeof(float));
  hipMemcpy(d_img_origin, h_img_origin, 3 * sizeof(float), hipMemcpyHostToDevice);

  std::vector<float> h_img = readArrayFromFile<float>("img.txt");
  float *d_img;
  hipMalloc(&d_img, h_img.size() * sizeof(float));
  hipMemcpy(d_img, h_img.data(), h_img.size() * sizeof(float), hipMemcpyHostToDevice);

  std::vector<float> h_vstart = readArrayFromFile<float>("vstart.txt");
  float *d_vstart;
  hipMalloc(&d_vstart, h_vstart.size() * sizeof(float));
  hipMemcpy(d_vstart, h_vstart.data(), h_vstart.size() * sizeof(float), hipMemcpyHostToDevice);

  std::vector<float> h_vend = readArrayFromFile<float>("vend.txt");
  float *d_vend;
  hipMalloc(&d_vend, h_vend.size() * sizeof(float));
  hipMemcpy(d_vend, h_vend.data(), h_vend.size() * sizeof(float), hipMemcpyHostToDevice);

  size_t nlors = h_vstart.size() / 3;

  float *d_xstart, *d_xend;
  hipMalloc(&d_xstart, 3 * nlors * sizeof(float));
  hipMalloc(&d_xend, 3 * nlors * sizeof(float));

  for (int ir = 0; ir < nlors; ir++)
  {
    for (int j = 0; j < 3; j++)
    {
      float xstart_val = h_img_origin[j] + h_vstart[ir * 3 + j] * h_voxsize[j];
      float xend_val = h_img_origin[j] + h_vend[ir * 3 + j] * h_voxsize[j];
      hipMemcpy(&d_xstart[ir * 3 + j], &xstart_val, sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(&d_xend[ir * 3 + j], &xend_val, sizeof(float), hipMemcpyHostToDevice);
    }
  }
  float *d_img_fwd;
  hipMalloc(&d_img_fwd, nlors * sizeof(float));
  joseph3d_fwd(d_xstart, d_xend, d_img, d_img_origin, d_voxsize, d_img_fwd, nvoxels, nlors, d_img_dim, device_id, threadsperblock);

  std::vector<float> h_img_fwd(nlors);
  hipMemcpy(h_img_fwd.data(), d_img_fwd, nlors * sizeof(float), hipMemcpyDeviceToHost);

  std::vector<float> h_expected_fwd_vals = readArrayFromFile<float>("expected_fwd_vals.txt");
  float fwd_diff = 0;
  float eps = 1e-7;

  for (int ir = 0; ir < nlors; ir++)
  {
    fwd_diff = std::abs(h_img_fwd[ir] - h_expected_fwd_vals[ir]);
    if (fwd_diff > eps)
    {
      std::cerr << "CUDA device array test failed for ray " << ir << "\n";
      return;
    }
  }

  // Test the back projection
  float *d_bimg;
  hipMalloc(&d_bimg, h_img_dim[0] * h_img_dim[1] * h_img_dim[2] * sizeof(float));
  hipMemset(d_bimg, 0, h_img_dim[0] * h_img_dim[1] * h_img_dim[2] * sizeof(float));

  float *d_ones;
  hipMalloc(&d_ones, nlors * sizeof(float));
  std::vector<float> h_ones(nlors, 1.0f);
  hipMemcpy(d_ones, h_ones.data(), nlors * sizeof(float), hipMemcpyHostToDevice);

  joseph3d_back(d_xstart, d_xend, d_bimg, d_img_origin, d_voxsize, d_ones, nvoxels, nlors, d_img_dim, device_id, threadsperblock);

  std::vector<float> h_bimg(h_img_dim[0] * h_img_dim[1] * h_img_dim[2]);
  hipMemcpy(h_bimg.data(), d_bimg, h_bimg.size() * sizeof(float), hipMemcpyDeviceToHost);

  printf("\nCUDA device back projection of ones along all rays:\n");
  for (size_t i0 = 0; i0 < h_img_dim[0]; i0++)
  {
    for (size_t i1 = 0; i1 < h_img_dim[1]; i1++)
    {
      for (size_t i2 = 0; i2 < h_img_dim[2]; i2++)
      {
        printf("%.1f ", h_bimg[h_img_dim[1] * h_img_dim[2] * i0 + h_img_dim[2] * i1 + i2]);
      }
      printf("\n");
    }
    printf("\n");
  }

  // Validate the back projection using adjointness
  float inner_product1 = 0.0f;
  float inner_product2 = 0.0f;

  for (size_t i = 0; i < h_img.size(); i++)
  {
    inner_product1 += h_img[i] * h_bimg[i];
  }

  for (size_t ir = 0; ir < nlors; ir++)
  {
    inner_product2 += h_img_fwd[ir] * h_ones[ir];
  }

  float ip_diff = fabs(inner_product1 - inner_product2);

  if (ip_diff > eps)
  {
    std::cerr << "CUDA device array back projection test failed: adjointness property violated.\n";
  }

  hipFree(d_img_dim);
  hipFree(d_voxsize);
  hipFree(d_img);
  hipFree(d_vstart);
  hipFree(d_vend);
  hipFree(d_xstart);
  hipFree(d_xend);
  hipFree(d_img_fwd);
  hipFree(d_bimg);
  hipFree(d_ones);
}

void test_box_projection_cuda_managed_arrays(int device_id, int threadsperblock)
{
  hipSetDevice(device_id);

  // Test parameters
  std::vector<float> voxel_size = {2.0f, 1.0f, 4.0f};
  std::vector<int> img_dim = {50, 100, 25};
  std::vector<float> img_origin = {-50.0f + 0.5f * voxel_size[0],
                                   -50.0f + 0.5f * voxel_size[1],
                                   -50.0f + 0.5f * voxel_size[2]};
  size_t nvox = img_dim[0] * img_dim[1] * img_dim[2];

  std::vector<std::vector<float>> xstart = {
      {100, 0, 0}, {50, 0, 0}, {0, 50, 0}, {0, 0, 50}, {40, 0, 0}, {0, 40, 0}, {0, 0, 40}, {50, 5, 0}, {0, 50, 5}, {5, 0, 50}, {50, 5, -2}, {-2, 50, 5}, {5, -2, 50}};
  std::vector<std::vector<float>> xend = {
      {-100, 0, 0}, {-50, 0, 0}, {0, -50, 0}, {0, 0, -50}, {-40, 0, 0}, {0, -40, 0}, {0, 0, -40}, {-50, -4, 0}, {0, -50, -4}, {-4, 0, -50}, {-50, -4, 3}, {3, -50, -4}, {-4, 3, -50}};

  float sqrt_100_9 = static_cast<float>(std::sqrt(100 * 100 + 9 * 9));
  float sqrt_100_9_5 = static_cast<float>(std::sqrt(100 * 100 + 9 * 9 + 5 * 5));

  std::vector<float> exp_vals = {
      100.0f, 100.0f, 100.0f, 100.0f,
      80.0f, 80.0f, 80.0f,
      sqrt_100_9, sqrt_100_9, sqrt_100_9,
      sqrt_100_9_5, sqrt_100_9_5, sqrt_100_9_5};

  size_t n_lors = xstart.size();
  std::vector<float> xstart_flat, xend_flat;
  for (size_t i = 0; i < n_lors; ++i)
  {
    xstart_flat.insert(xstart_flat.end(), xstart[i].begin(), xstart[i].end());
    xend_flat.insert(xend_flat.end(), xend[i].begin(), xend[i].end());
  }

  // Managed allocations
  float *d_xstart, *d_xend, *d_img, *d_img_origin, *d_voxel_size, *d_img_fwd;
  int *d_img_dim;
  hipMallocManaged(&d_xstart, xstart_flat.size() * sizeof(float));
  hipMallocManaged(&d_xend, xend_flat.size() * sizeof(float));
  hipMallocManaged(&d_img, nvox * sizeof(float));
  hipMallocManaged(&d_img_origin, 3 * sizeof(float));
  hipMallocManaged(&d_voxel_size, 3 * sizeof(float));
  hipMallocManaged(&d_img_fwd, n_lors * sizeof(float));
  hipMallocManaged(&d_img_dim, 3 * sizeof(int));

  std::copy(xstart_flat.begin(), xstart_flat.end(), d_xstart);
  std::copy(xend_flat.begin(), xend_flat.end(), d_xend);
  std::fill_n(d_img, nvox, 1.0f);
  std::copy(img_origin.begin(), img_origin.end(), d_img_origin);
  std::copy(voxel_size.begin(), voxel_size.end(), d_voxel_size);
  std::copy(img_dim.begin(), img_dim.end(), d_img_dim);

  joseph3d_fwd(
      d_xstart, d_xend, d_img, d_img_origin, d_voxel_size, d_img_fwd,
      nvox, n_lors, d_img_dim, device_id, threadsperblock);

  hipDeviceSynchronize();

  float eps = 1e-4f;
  for (size_t i = 0; i < n_lors; ++i)
  {
    if (std::abs(d_img_fwd[i] - exp_vals[i]) >= eps)
    {
      std::cerr << "[Managed] Forward box projection test failed at i=" << i
                << ": got " << d_img_fwd[i] << ", expected " << exp_vals[i] << std::endl;
    }
  }

  hipFree(d_xstart);
  hipFree(d_xend);
  hipFree(d_img);
  hipFree(d_img_origin);
  hipFree(d_voxel_size);
  hipFree(d_img_fwd);
  hipFree(d_img_dim);
}

void test_box_projection_cuda_device_arrays(int device_id, int threadsperblock)
{
  hipSetDevice(device_id);

  // Test parameters
  std::vector<float> voxel_size = {2.0f, 1.0f, 4.0f};
  std::vector<int> img_dim = {50, 100, 25};
  std::vector<float> img_origin = {-50.0f + 0.5f * voxel_size[0],
                                   -50.0f + 0.5f * voxel_size[1],
                                   -50.0f + 0.5f * voxel_size[2]};
  size_t nvox = img_dim[0] * img_dim[1] * img_dim[2];

  std::vector<std::vector<float>> xstart = {
      {100, 0, 0}, {50, 0, 0}, {0, 50, 0}, {0, 0, 50}, {40, 0, 0}, {0, 40, 0}, {0, 0, 40}, {50, 5, 0}, {0, 50, 5}, {5, 0, 50}, {50, 5, -2}, {-2, 50, 5}, {5, -2, 50}};
  std::vector<std::vector<float>> xend = {
      {-100, 0, 0}, {-50, 0, 0}, {0, -50, 0}, {0, 0, -50}, {-40, 0, 0}, {0, -40, 0}, {0, 0, -40}, {-50, -4, 0}, {0, -50, -4}, {-4, 0, -50}, {-50, -4, 3}, {3, -50, -4}, {-4, 3, -50}};

  float sqrt_100_9 = static_cast<float>(std::sqrt(100 * 100 + 9 * 9));
  float sqrt_100_9_5 = static_cast<float>(std::sqrt(100 * 100 + 9 * 9 + 5 * 5));

  std::vector<float> exp_vals = {
      100.0f, 100.0f, 100.0f, 100.0f,
      80.0f, 80.0f, 80.0f,
      sqrt_100_9, sqrt_100_9, sqrt_100_9,
      sqrt_100_9_5, sqrt_100_9_5, sqrt_100_9_5};

  size_t n_lors = xstart.size();
  std::vector<float> xstart_flat, xend_flat;
  for (size_t i = 0; i < n_lors; ++i)
  {
    xstart_flat.insert(xstart_flat.end(), xstart[i].begin(), xstart[i].end());
    xend_flat.insert(xend_flat.end(), xend[i].begin(), xend[i].end());
  }

  // Device allocations
  float *dxstart, *dxend, *dimg, *dimg_origin, *dvoxel_size, *dimg_fwd;
  int *dimg_dim;
  hipMalloc(&dxstart, xstart_flat.size() * sizeof(float));
  hipMalloc(&dxend, xend_flat.size() * sizeof(float));
  hipMalloc(&dimg, nvox * sizeof(float));
  hipMalloc(&dimg_origin, 3 * sizeof(float));
  hipMalloc(&dvoxel_size, 3 * sizeof(float));
  hipMalloc(&dimg_fwd, n_lors * sizeof(float));
  hipMalloc(&dimg_dim, 3 * sizeof(int));

  hipMemcpy(dxstart, xstart_flat.data(), xstart_flat.size() * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dxend, xend_flat.data(), xend_flat.size() * sizeof(float), hipMemcpyHostToDevice);
  std::vector<float> ones_img(nvox, 1.0f);
  hipMemcpy(dimg, ones_img.data(), nvox * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dimg_origin, img_origin.data(), 3 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dvoxel_size, voxel_size.data(), 3 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dimg_dim, img_dim.data(), 3 * sizeof(int), hipMemcpyHostToDevice);

  joseph3d_fwd(
      dxstart, dxend, dimg, dimg_origin, dvoxel_size, dimg_fwd,
      nvox, n_lors, dimg_dim, device_id, threadsperblock);

  std::vector<float> img_fwd2(n_lors, 0.0f);
  hipMemcpy(img_fwd2.data(), dimg_fwd, n_lors * sizeof(float), hipMemcpyDeviceToHost);

  float eps = 1e-4f;
  for (size_t i = 0; i < n_lors; ++i)
  {
    if (std::abs(img_fwd2[i] - exp_vals[i]) >= eps)
    {
      std::cerr << "[Device] Forward box projection test failed at i=" << i
                << ": got " << img_fwd2[i] << ", expected " << exp_vals[i] << std::endl;
    }
  }

  hipFree(dxstart);
  hipFree(dxend);
  hipFree(dimg);
  hipFree(dimg_origin);
  hipFree(dvoxel_size);
  hipFree(dimg_fwd);
  hipFree(dimg_dim);
}
